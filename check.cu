
#include <hip/hip_runtime.h>
#include <stdio.h>

const int WARP_SIZE = 32;

__global__ void check(float* a, float* total)
{
  __shared__ float values[WARP_SIZE];

  values[threadIdx.x] = a[threadIdx.x];
  __syncthreads();

  float val = a[threadIdx.x];

  if (threadIdx.x == 1)
    for (int i = 0; i < WARP_SIZE; i++)
    {
      float v1 =  __shfl(val, i);
      if (v1 != values[i])
        atomicAdd(total, 1);
    }
}


int main()
{
  float* data;
  hipMallocManaged(&data, WARP_SIZE * sizeof(float));
  for (int i = 0; i < WARP_SIZE; i++)
    data[i] = i + 1;

  float* total;
  hipMallocManaged(&total, sizeof(float));
  *total = 0.0;

  check<<<1, 32>>>(data, total);
  hipDeviceSynchronize();

  printf("check: %.1lf\n", *total);

  hipFree(total);
  hipFree(data);
  return 0;
}
	
